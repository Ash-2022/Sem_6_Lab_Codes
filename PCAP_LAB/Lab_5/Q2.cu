#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

__global__ void addVector(int*a , int*b , int*c , int size){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < size) c[id] = a[id] + b[id];
}

const int NUM_THREADS_PER_BLK = 256;

int main(void){
    // Allocate Host memory
    int size;
    printf("Enter Num of elements of vector : ");
    scanf("%d" , &size);
    int* vector1 = (int*) malloc(size * sizeof(int));
    int* vector2 = (int*) malloc(size * sizeof(int));
    int* vector3 = (int*) malloc(size * sizeof(int));
    if(size < 5){
        printf("Vector 1 : \n");
        for(int i = 0; i < size; i++){
            printf("Enter ele %d : " , (i+1));
            scanf("%d" , &vector1[i]);
        }
        printf("Vector 2 : \n");
        for(int i = 0; i < size; i++){
            printf("Enter ele %d : " , (i+1));
            scanf("%d" , &vector2[i]);
        }
    }
    else{
        for(int i = 0; i < size; i++){
            vector1[i] = i;
            vector2[i] = size + i;
        }
    }
    int vectorSize = size * sizeof(int);

    //Allocate Device Memory
    int * d_vector1 , *d_vector2 , *d_vector3;
    hipMalloc((void**) &d_vector1 , vectorSize);
    hipMalloc((void**) &d_vector2 , vectorSize);
    hipMalloc((void**) &d_vector3 , vectorSize);

    // Copy inputs to device
    hipMemcpy(d_vector1 , vector1 , vectorSize , hipMemcpyHostToDevice);
    hipMemcpy(d_vector2 , vector2 , vectorSize , hipMemcpyHostToDevice);
    hipMemcpy(d_vector3 , vector3 , vectorSize , hipMemcpyHostToDevice);

    // Block Creation
    int numBlks = size / NUM_THREADS_PER_BLK + 1;

    //Launch add()kernal on gpu
    addVector<<<numBlks , NUM_THREADS_PER_BLK>>>(d_vector1 , d_vector2 , d_vector3 , size);
    
    //Copy result back to host
    hipMemcpy(vector3 , d_vector3 , vectorSize , hipMemcpyDeviceToHost);
    
    //Print result
    printf("Vector 3 : %d blocks of 256 threads\n" , numBlks);
    for(int i = 0; i < size;i++){
        printf("%d " , vector3[i]);
    }
    printf("\n");

    //Cleanup
    hipFree(d_vector1);
    hipFree(d_vector2);
    hipFree(d_vector3);
    free(vector1);
    free(vector2);
    free(vector3);
}