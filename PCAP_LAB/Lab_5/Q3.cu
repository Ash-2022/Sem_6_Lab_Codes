#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

__global__ void calcSine(double*a , double*c , int size){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < size) c[id] = sin(a[id]);
}

const int NUM_THREADS_PER_BLK = 64;
const double PI = 3.1414;

int main(void){
    // Allocate Host memory
    int size;
    printf("Enter Num of elements of vector : ");
    scanf("%d" , &size);
    double* vector1 = (double*) malloc(size * sizeof(double));
    double* vector2 = (double*) malloc(size * sizeof(double));
    for(int i = 0; i < size; i++){
        vector1[i] = (i/(double)180) * PI;
    }
    int vectorSize = size * sizeof(double);

    //Allocate Device Memory
    double * d_vector1 , *d_vector2;
    hipMalloc((void**) &d_vector1 , vectorSize);
    hipMalloc((void**) &d_vector2 , vectorSize);

    // Copy inputs to device
    hipMemcpy(d_vector1 , vector1 , vectorSize , hipMemcpyHostToDevice);
    hipMemcpy(d_vector2 , vector2 , vectorSize , hipMemcpyHostToDevice);

    // Block Creation
    int numBlks = size / NUM_THREADS_PER_BLK + 1;

    //Launch add()kernal on gpu
    calcSine<<<numBlks , NUM_THREADS_PER_BLK>>>(d_vector1 , d_vector2 , size);
    
    //Copy result back to host
    hipMemcpy(vector2 , d_vector2 , vectorSize , hipMemcpyDeviceToHost);
    
    //Print result
    printf("Sine Vector : %d blocks of 256 threads\n" , numBlks);
    for(int i = 0; i < size;i++){
        printf("%lf " , vector2[i]);
    }
    printf("\n");

    //Cleanup
    hipFree(d_vector1);
    hipFree(d_vector2);
    free(vector1);
    free(vector2);
}