#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void add(int*a , int*b , int*c){
    *c = *a + *b;
}

int main(void){
    int a , b , c;
    int *d_a , *d_b , *d_c;
    int size = sizeof(int);

    // Allocate Space for device copies
    hipMalloc((void**) &d_a , size);
    hipMalloc((void**) &d_b , size);
    hipMalloc((void**) &d_c , size);
    //Setup Vals
    a = 3;
    b= 5;
    // Copy inputs to device
    hipMemcpy(d_a , &a , size , hipMemcpyHostToDevice);
    hipMemcpy(d_b , &b , size , hipMemcpyHostToDevice);
    hipMemcpy(d_c , &c , size , hipMemcpyHostToDevice);
    // Launch add() kernal on gpu
    add<<<1,1>>>(d_a , d_b , d_c);
    //COpy result back to host
    hipMemcpy(&c , d_c , size , hipMemcpyDeviceToHost);
    printf("Result : %d\n" , c);
    //Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}