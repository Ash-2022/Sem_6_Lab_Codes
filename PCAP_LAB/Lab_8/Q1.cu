#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

const int MAX_BLK_THREADS = 1024;

__global__ void addMatrixElementWise(int * A , int * B , int * C , int r , int c){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < (r*c)){
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void addMatrixRowWise(int * A , int * B , int * C){
    int r = blockIdx.y;
    int c = blockDim.x;
    for(int i = 0; i < c; i++){
        C[r*c + i] = A[r*c + i] + B[r*c + i];
    }
}

__global__ void addMatrixColWise(int * A , int * B , int * C , int m , int n){
    int r = threadIdx.x; 
    int c = blockIdx.x; 
    if (r < m) {
        C[r * n + c] = A[r * n + c] + B[r * n + c]; 
    }
}

void displayMatrix(int * mat , int m , int n){
    for(int i = 0; i < m*n; i++){
        if(!(i % m)) printf("\n");
        printf("%d " , mat[i]);
    }
    printf("\n");
}

int main(){
    // Host Allocation
    int m , n;
    int *matA , *matB , *matC , *matD , *matE;

    printf("Enter Size of matrix : (Rows , Col) : ");
    scanf("%d %d" , &m , &n);
    int matSize = m * n * sizeof(int);
    
    matA = (int*) malloc(matSize);
    matB = (int*) malloc(matSize);
    matC = (int*) malloc(matSize);
    matD = (int*) malloc(matSize);
    matE = (int*) malloc(matSize);

    // Rand Initialization
    for(int i = 0; i < m * n;i++){
        matA[i] = rand() % 100;
        matB[i] = rand() % 100;
        matC[i] = 0;
        matD[i] = 0;
        matE[i] = 0;
    }

    // Display Host Matricies
    printf("\nMatrix A :");
    displayMatrix(matA , m , n);
    
    printf("\nMatrix B :");
    displayMatrix(matB , m , n);

    // Device Allocation
    int *d_matA , *d_matB , *d_matC , *d_matD , *d_matE;
    hipMalloc(&d_matA , matSize);
    hipMalloc(&d_matB , matSize);
    hipMalloc(&d_matC , matSize);
    hipMalloc(&d_matD , matSize);
    hipMalloc(&d_matE , matSize);

    // Host To Device Copy
    hipMemcpy(d_matA , matA , matSize , hipMemcpyHostToDevice);
    hipMemcpy(d_matB , matB , matSize , hipMemcpyHostToDevice);
    hipMemcpy(d_matC , matC , matSize , hipMemcpyHostToDevice);
    hipMemcpy(d_matD , matD , matSize , hipMemcpyHostToDevice);
    hipMemcpy(d_matE , matE , matSize , hipMemcpyHostToDevice);

    // Element Wise Kernal Setup
    int numBlks = (m * n + MAX_BLK_THREADS - 1) / MAX_BLK_THREADS;
    dim3 gridElementWise(numBlks , 1 , 1);
    dim3 blkElementWise(MAX_BLK_THREADS , 1 , 1);
    addMatrixElementWise<<<gridElementWise,blkElementWise>>>(d_matA , d_matB , d_matC , m , n);
    hipMemcpy(matC , d_matC , matSize , hipMemcpyDeviceToHost);
    hipFree(d_matC);
    
    // Row Wise Kernal Setup
    dim3 gridRowWise(1 , m , 1);
    dim3 blkRowWise(n , 1 , 1);
    addMatrixRowWise<<<gridRowWise,blkRowWise>>>(d_matA , d_matB , d_matD);
    hipMemcpy(matD , d_matD , matSize , hipMemcpyDeviceToHost);
    hipFree(d_matD);

    // Col Wise Kernal Setup
    dim3 gridColWise(n, 1, 1);   
    dim3 blkColWise(m, 1, 1); 

    addMatrixColWise<<<gridColWise, blkColWise>>>(d_matA, d_matB, d_matE, m, n);
    hipMemcpy(matE , d_matE , matSize , hipMemcpyDeviceToHost);
    hipFree(d_matE);

    // Device Memory Free
    hipFree(d_matA);
    hipFree(d_matB);

    // Display Result
    printf("\nMatrix C (Element Wise) :");
    displayMatrix(matC , m , n);

    printf("\nMatrix D (Row Wise) :");
    displayMatrix(matD , m , n);

    printf("\nMatrix E (Col Wise) :");
    displayMatrix(matE , m , n);

    // Host Memory Free
    free(matA);
    free(matB);
    free(matC);
    free(matD);
    free(matE);

    return 0;
}